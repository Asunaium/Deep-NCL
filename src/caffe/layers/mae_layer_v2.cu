#include "hip/hip_runtime.h"
#include <vector>
#include <math.h>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/mae_loss_layer_v2.hpp"

namespace caffe {

template <typename Dtype>
void MAELossV2Layer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  caffe_gpu_mul(
      count,
      bottom[0]->gpu_data(),
      bottom[2]->gpu_data(),
      multi1_.mutable_gpu_data());
  caffe_gpu_mul(
      count,
      bottom[1]->gpu_data(),
      bottom[2]->gpu_data(),
      multi2_.mutable_gpu_data());
  Dtype asum1,asum2;
  caffe_gpu_asum(count, multi1_.gpu_data(), &asum1);
  caffe_gpu_asum(count, multi2_.gpu_data(), &asum2);
  Dtype loss = abs(asum1-asum2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void MAELossV2Layer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
}

INSTANTIATE_LAYER_GPU_FUNCS(MAELossV2Layer);

}  // namespace caffe
