#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/ncl_loss_layer.hpp"

namespace caffe {

template <typename Dtype>
void NCLLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  Dtype lambda = this->layer_param_.ncl_loss_param().lambda();
  int net_num = this->layer_param_.ncl_loss_param().net_num();
  Dtype norm = Dtype(1)/(Dtype(net_num)*Dtype(net_num)-lambda*(Dtype(net_num)-Dtype(1))*(Dtype(net_num)-Dtype(1)));
  caffe_gpu_sub(
      count,
      bottom[2]->gpu_data(),
      bottom[0]->gpu_data(),
      avgdiff_.mutable_gpu_data());

  caffe_gpu_scal(count,lambda*(Dtype(net_num)-Dtype(1)),avgdiff_.mutable_gpu_data());
  caffe_gpu_axpby(
          count,              // count
          Dtype(net_num)*Dtype(net_num),                              // alpha
          bottom[1]->gpu_data(),                   // a
          Dtype(0),                           // beta
          label_.mutable_gpu_data());  // b

  caffe_gpu_sub(
      count,
      label_.gpu_data(),
      avgdiff_.gpu_data(),
      label_.mutable_gpu_data());
  caffe_gpu_scal(count,norm,label_.mutable_gpu_data());
  
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      label_.gpu_data(),
      diff_.mutable_gpu_data());

  Dtype dot;
  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  Dtype loss = dot / bottom[0]->num() / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void NCLLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[i]->num();
      caffe_gpu_axpby(
          bottom[i]->count(),              // count
          alpha,                              // alpha
          diff_.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[i]->mutable_gpu_diff());  // b
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(NCLLossLayer);

}  // namespace caffe
